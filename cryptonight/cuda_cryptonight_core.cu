#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cryptonight.h"

#ifndef _WIN32
#include <unistd.h>
#endif

extern int device_arch[8][2];
extern int device_bfactor[8];
extern int device_bsleep[8];

#include "cuda_cryptonight_aes.cu"

__device__ __forceinline__ uint64_t cuda_mul128(uint64_t multiplier, uint64_t multiplicand, uint64_t* product_hi)
{
    *product_hi = __umul64hi(multiplier, multiplicand);
    return(multiplier * multiplicand);
}

__global__ void cryptonight_core_gpu_phase1(int threads, uint32_t *long_state, struct cryptonight_gpu_ctx *ctx)
{
	__shared__ uint32_t sharedMemory[1024];

    cn_aes_gpu_init(sharedMemory);

	__syncthreads();

    int thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 3;
    int sub = (threadIdx.x & 7) << 2;
   
    if (thread < threads)
    {
        uint32_t key[40], text[4], i;

        MEMCPY8(key, ctx[thread].key1, 20);
        MEMCPY8(text, &ctx[thread].state[sub + 16], 2);

        for(i = 0; i < 0x80000; i += 32)
        {
            cn_aes_pseudo_round_mut(sharedMemory, text, key);
            MEMCPY8(&long_state[(thread << 19) + sub + i], text, 2);
        }
    }
}

__global__ void cryptonight_core_gpu_phase2(int threads, int bfactor, int partidx, uint32_t *d_long_state, struct cryptonight_gpu_ctx *d_ctx)
{
	__shared__ uint32_t sharedMemory[1024];

    cn_aes_gpu_init(sharedMemory);

	__syncthreads();

#if __CUDA_ARCH__ >= 300

    int thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 2;
    int sub = threadIdx.x & 3;

    if (thread < threads)
    {
        int i, j, k;
        int batchsize = ITER >> (2+bfactor);
        int start = partidx * batchsize;
        int end = start + batchsize;
        uint32_t * __restrict__ long_state = &d_long_state[thread << 19];
        struct cryptonight_gpu_ctx *ctx = &d_ctx[thread];
        uint32_t a, b, c, x[4];
        uint32_t t1[4], t2[4], res;
        uint64_t reshi, reslo;

        a = ctx->a[sub];
        b = ctx->b[sub];

        for (i = start; i < end; ++i) {

            //j = ((uint32_t *)a)[0] & 0x1FFFF0;
            j = (__shfl((int)a, 0, 4) & 0x1FFFF0) >> 2;
            
            //cn_aes_single_round(sharedMemory, &long_state[j], c, a);
            x[0] = long_state[j + sub];
            x[1] = __shfl((int)x[0], sub+1, 4);
            x[2] = __shfl((int)x[0], sub+2, 4);
            x[3] = __shfl((int)x[0], sub+3, 4);
            c = a ^
                t_fn0(x[0] & 0xff) ^
                t_fn1((x[1] >> 8) & 0xff) ^
                t_fn2((x[2] >> 16) & 0xff) ^
                t_fn3((x[3] >> 24) & 0xff);
            
            //XOR_BLOCKS_DST(c, b, &long_state[j]);
            long_state[j + sub] = c ^ b;

            //MUL_SUM_XOR_DST(c, a, &long_state[((uint32_t *)c)[0] & 0x1FFFF0]);
            j = (__shfl((int)c, 0, 4) & 0x1FFFF0) >> 2;
            for( k = 0; k < 2; k++ ) t1[k] = __shfl((int)c, k, 4);
            for( k = 0; k < 4; k++ ) t2[k] = __shfl((int)a, k, 4);
            asm(
                "mad.lo.u64 %0, %2, %3, %4;\n\t"
                "mad.hi.u64 %1, %2, %3, %5;\n\t"
                : "=l"(reslo), "=l"(reshi)
                : "l"(((uint64_t *)t1)[0]), "l"(((uint64_t *)long_state)[j >> 1]), "l"(((uint64_t *)t2)[1]), "l"(((uint64_t *)t2)[0]));
            res = (sub & 2 ? reslo : reshi) >> (sub&1 ? 32 : 0);
            a = long_state[j + sub] ^ res;
            long_state[j + sub] = res;

            //j = ((uint32_t *)a)[0] & 0x1FFFF0;
            j = (__shfl((int)a, 0, 4) & 0x1FFFF0) >> 2;
            
            //cn_aes_single_round(sharedMemory, &long_state[j], b, a);
            x[0] = long_state[j + sub];
            x[1] = __shfl((int)x[0], sub+1, 4);
            x[2] = __shfl((int)x[0], sub+2, 4);
            x[3] = __shfl((int)x[0], sub+3, 4);
            b = a ^
                t_fn0(x[0] & 0xff) ^
                t_fn1((x[1] >> 8) & 0xff) ^
                t_fn2((x[2] >> 16) & 0xff) ^
                t_fn3((x[3] >> 24) & 0xff);

            //XOR_BLOCKS_DST(b, c, &long_state[j]);
            long_state[j + sub] = c ^ b;

            //MUL_SUM_XOR_DST(b, a, &long_state[((uint32_t *)b)[0] & 0x1FFFF0]);
            j = (__shfl((int)b, 0, 4) & 0x1FFFF0) >> 2;
            for( k = 0; k < 2; k++ ) t1[k] = __shfl((int)b, k, 4);
            for( k = 0; k < 4; k++ ) t2[k] = __shfl((int)a, k, 4);
            asm(
                "mad.lo.u64 %0, %2, %3, %4;\n\t"
                "mad.hi.u64 %1, %2, %3, %5;\n\t"
                : "=l"(reslo), "=l"(reshi)
                : "l"(((uint64_t *)t1)[0]), "l"(((uint64_t *)long_state)[j >> 1]), "l"(((uint64_t *)t2)[1]), "l"(((uint64_t *)t2)[0]));
            res = (sub & 2 ? reslo : reshi) >> (sub&1 ? 32 : 0);
            a = long_state[j + sub] ^ res;
            long_state[j + sub] = res;
        }

        if( bfactor > 0 ) {

            ctx->a[sub] = a;
            ctx->b[sub] = b;
        }
    }

#else // __CUDA_ARCH__ < 300

    int thread = blockDim.x * blockIdx.x + threadIdx.x;

    if (thread < threads)
    {
        int i, j;
        int batchsize = ITER >> (2+bfactor);
        int start = partidx * batchsize;
        int end = start + batchsize;
        uint32_t *long_state = &d_long_state[thread << 19];
        struct cryptonight_gpu_ctx *ctx = &d_ctx[thread];
        uint32_t a[4], b[4], c[4];

        MEMCPY8(a, ctx->a, 2);
        MEMCPY8(b, ctx->b, 2);

        for (i = start; i < end; ++i) {

            j = (a[0] & 0x1FFFF0) >> 2;
            cn_aes_single_round(sharedMemory, (uint8_t *)&long_state[j], c, a);
            XOR_BLOCKS_DST(c, b, &long_state[j]);
            MUL_SUM_XOR_DST(c, a, (uint8_t *)&long_state[(c[0] & 0x1FFFF0) >> 2]);
            j = (a[0] & 0x1FFFF0) >> 2;
            cn_aes_single_round(sharedMemory, (uint8_t *)&long_state[j], b, a);
            XOR_BLOCKS_DST(b, c, &long_state[j]);
            MUL_SUM_XOR_DST(b, a, &long_state[(b[0] & 0x1FFFF0) >> 2]);
        }
        
        if( bfactor > 0 ) {

            MEMCPY8(ctx->a, a, 2);
            MEMCPY8(ctx->b, b, 2);
        }
    }

#endif // __CUDA_ARCH__ >= 300
}

__global__ void cryptonight_core_gpu_phase3(int threads, uint32_t *long_state, struct cryptonight_gpu_ctx *ctx)
{
	__shared__ uint32_t sharedMemory[1024];

    cn_aes_gpu_init(sharedMemory);

	__syncthreads();

    int thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 3;
    int sub = (threadIdx.x & 7) << 2;
   
    if (thread < threads)
    {
        uint32_t key[40], text[4], i, j;
        MEMCPY8(key, ctx[thread].key2, 20);
        MEMCPY8(text, &ctx[thread].state[sub + 16], 2);

        for (i = 0; i < 0x80000; i += 32)
        {
            for(j = 0; j < 4; ++j)
                text[j] ^= long_state[(thread << 19) + sub + i + j];

            cn_aes_pseudo_round_mut(sharedMemory, text, key);
        }
        
        MEMCPY8(&ctx[thread].state[sub + 16], text, 2);
    }
}


__host__ void cryptonight_core_cpu_init(int thr_id, int threads)
{
	cn_aes_cpu_init();
}

__host__ void cryptonight_core_cpu_hash(int thr_id, int blocks, int threads, uint32_t *d_long_state, struct cryptonight_gpu_ctx *d_ctx)
{
    dim3 grid(blocks);
    dim3 block(threads);
    dim3 block4(threads << 2);
    dim3 block8(threads << 3);

    size_t shared_size = 1024;
    int i, partcount = 1 << device_bfactor[thr_id];

    cryptonight_core_gpu_phase1<<<grid, block8, shared_size>>>(blocks*threads, d_long_state, d_ctx);
    hipDeviceSynchronize();
    if( partcount > 1 ) usleep(device_bsleep[thr_id]);

    for( i = 0; i < partcount; i++ ) {
        cryptonight_core_gpu_phase2<<<grid, (device_arch[thr_id][0] >= 3 ? block4 : block), shared_size>>>(blocks*threads, device_bfactor[thr_id], i, d_long_state, d_ctx);
        hipDeviceSynchronize();
        if( partcount > 1 ) usleep(device_bsleep[thr_id]);
    }

    cryptonight_core_gpu_phase3<<<grid, block8, shared_size>>>(blocks*threads, d_long_state, d_ctx);
    hipDeviceSynchronize();
}
